#include "hip/hip_runtime.h"
﻿/* This program will do a matrix addition on two matrices A and B.
They have the same dimension N (rows) x M (cols) where
N is 2048 and M is 1024.

+---------+   +---------+   +---------+
|000000000|   |111111111|   |111111111|
|111111111| + |222222222| = |333333333|
|222222222|   |333333333|   |555555555|
+---------+   +---------+   +---------+
matrixA       matrixB

matrixA   = column number starting at 0
matrixB   = column number + 1 starting at 0

Implement a 2D Grid-Stride-Loop:
https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
*/

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <stdio.h>

using namespace std;


/* This is our CUDA call wrapper, we will use in PAC.
*
*  Almost all CUDA calls should be wrapped with this makro.
*  Errors from these calls will be catched and printed on the console.
*  If an error appears, the program will terminate.
*
* Example: gpuErrCheck(hipMalloc(&deviceA, N * sizeof(int)));
*          gpuErrCheck(hipMemcpy(deviceA, hostA, N * sizeof(int), hipMemcpyHostToDevice));
*/
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        std::cout << "GPUassert: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
        if (abort)
        {
            exit(code);
        }
    }
}


// Compare results
int compareResultVec(int* matCPU, int* matGPU, int size)
{
    int error = 0;
    for (int i = 0; i < size; i++)
    {
        error += abs(matCPU[i] - matGPU[i]);
    }
    if (error == 0)
    {
        cout << "No errors. All good!" << endl;
        return 0;
    }
    else
    {
        cout << "Accumulated error: " << error << endl;
        return -1;
    }
}


void main(void)
{
    // Define the size of the matrix: M=1024 and N=2048
    const int M = 1 << 10;
    const int N = M << 1;

    //Please implement me
}
